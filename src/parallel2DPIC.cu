/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <math.h>
#include <time.h>
#include <fstream>
#include <vector>
#include <fstream>
#include <hiprand/hiprand_kernel.h>
#include <hipfft/hipfft.h>
#include <complex.h>
#include "float.h"

float L,LL;

int N, C,itera;

using namespace std;

// función Maxwelliana de la distribución de las partículas.
__device__ float distribution (float vb, float aleatorio, hiprandState *states)     //generador de distribución maxwelliana para la velocidad
{

  // Genera un valor random v
   float fmax = 0.5 * (1.0 + exp (-2.0 * vb * vb));
   float vmin = - 5.0 * vb;
   float vmax = + 5.0 * vb;
   float v;
   float f;
   float x;
   int Idx = blockIdx.x*blockDim.x + threadIdx.x;

   while(true){
	   v = vmin + ((vmax - vmin) * aleatorio);
	   f = 0.5 * (exp (-(v - vb) * (v - vb) / 2.0) +
			    exp (-(v + vb) * (v + vb) / 2.0));
	   x = fmax * aleatorio;
	   if(x > f) aleatorio = hiprand_uniform(states + Idx);
	   else return v;
   }

 return 0;
}
//Distribución aleatoria de las partículas.
__global__ void distribucionParticulas(float *rx,float *ry,float *vx,float *vy,int N,hiprandState *states,float vb,float L, int seed){
	int Idx = blockIdx.x*blockDim.x + threadIdx.x;

	seed = (unsigned int) (clock() * Idx);
	hiprand_init(seed, 0, 0, states + Idx);

	if(Idx < N){
		rx[Idx] = L*hiprand_uniform(states + Idx);    //inicializando la posicion aleatoria en x
		ry[Idx] = L*hiprand_uniform(states + Idx);
		vx[Idx] = distribution(vb,hiprand_uniform(states + Idx),states);//;L*curand_uniform_float(states + Idx);//distribution(vb,states);                          //inicializa la velocidad con una distribucion maxwelliana
		vy[Idx] = distribution(vb,hiprand_uniform(states + Idx),states);//L*curand_uniform_float(states + Idx);//distribution(vb,states);                          //inicializa la velocidad con una distribucion maxwelliana

	}


}
// inicialización de la densidad.
__global__ void inicializacionDensidad(float *ne,int C){
	int Id=blockIdx.x*blockDim.x + threadIdx.x;
	if(Id<(C*C)){
		ne[Id] = 0.0;
	}
 }

__global__ void inicializacionValoresReales(float *vr,int C){
	int Id=blockIdx.x*blockDim.x + threadIdx.x;
	if(Id<(C*C)){
		vr[Id] = 0.0;
	}
 }

//Calculo de la densidad en cada celda.

__global__ void calculoDensidadInicializacionCeldas(float *rx, float *ry, int *jx,int *jy,float *yx, int N, int C,float L){
	int Id = blockIdx.x*blockDim.x + threadIdx.x;
	 float dx = L / float (C);
	 //float dxx = L /float(C*C);
	if(Id < N){
		 jx[Id] = int(rx[Id]/dx); //posicion en x de la particula
		 jy[Id] = int(ry[Id]/dx); //posicion en y de la particula
		 yx[Id] = (rx[Id]/dx) - (float)jx[Id]; //posicion exacta de la particula en x de la celda "j"
    }

}
__global__ void calculoDensidad(float *ne, int *jx, int *jy,float *yx, int C, float L, int N){
	 float dxx = L /float(C*C);
	// int Id = blockIdx.x*blockDim.x + threadIdx.x;
	 for(int i=0; i<N; i++){
		ne[(jy[i]*C)+jx[i]] += (1. - yx[i])/dxx;
		if(jx[i]+1 == C) ne[(jy[i]*C)] += yx[i]/dxx;
		else ne[(jy[i]*C)+jx[i]+1] += yx[i]/dxx;
	 }

}

__global__ void normalizacionDensidad(float *ne,float *n, int N, int C, float L){
	 int Id = blockIdx.x*blockDim.x + threadIdx.x;
	 if(Id<C*C){
		 n[Id]=float(C*C)*ne[Id]/float(N)-1;
	 }

}

// función que integra la densidad normalizada con la otra densidad
void Output(float *ne_d, float *n_d, int *jx_d,int *jy_d,float *yx_d,int C,float L,int N){
	//definicion de los bloques.
	float blockSize = 1024;
	dim3 dimBlock (ceil(N/blockSize), 1, 1);
	dim3 dimBlock2 (ceil(C*C/blockSize), 1, 1);
	dim3 dimGrid (blockSize, 1, 1);

	calculoDensidad<<<1,1>>>(ne_d,jx_d,jy_d,yx_d,C,L,N);//proceso de mejora.
	hipDeviceSynchronize();
	normalizacionDensidad<<<blockSize,dimBlock2>>>(ne_d,n_d,N,C,L);
	hipDeviceSynchronize();


}

//////////////////////////////////////////////////////////////////////////////////////////////////
//Calculo Poisson.

/* en este punto se asignan los valores de la densidad normalizada a una variable compleja que es la que entra a operar con la
 * transformada rápida de fourier en cufft.
 */


__global__ void realTocomplex(float *n_d, hipfftComplex *n_d_C, int C){
	int i= blockIdx.x*blockDim.x+threadIdx.x;
	//int j= blockIdx.y*blockDim.y+threadIdx.y;
	//int index= (i*C)+j;// recorrido de la matriz
	if(i<C*C){
		n_d_C[i].x = n_d[i];
		n_d_C[i].y = 0.0f;

	}
}
/////////////////////////////////////////////////////////////////////////////////////////////////////
/*Normalizacion de la transformada hacia adelante*/

__global__ void normalizacionSalidaTranfForward(hipfftComplex *T_F, hipfftComplex *T_F_N, int C){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	//int j = blockIdx.y*blockDim.y + threadIdx.y;
	//int index= i*C+j;
	  if(i<C*C){
		  T_F_N[i].x=T_F[i].x/float(C*C*C*C);
		  T_F_N[i].y=T_F[i].y/float(C*C*C*C);

	  }
}

/////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void ComplexToFloat2( hipfftComplex *T_F_N, float2 *poisson_d,  int C){

  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if(i<C*C){
	  poisson_d[i].x=T_F_N[i].x;
	  poisson_d[i].y =T_F_N[i].y;

	  //float(1.e+7);

  }
}

/////////////////////////////////////////////////////////////////////////////////////////////////////
///*Calculo Poisson*/

void Poisson(float2 *calculoPoisson_h, float L, int C){// pasar a calculo paralelo
	float dx = L / float (C);
	calculoPoisson_h[0].x=0.0;
	calculoPoisson_h[0].y=0.0;
	float2 i;
	i.x=0.0;
	i.y=L; //creamos una variable compleja para poder aplicar la discretizacion.
	float2 W;
	W.x= exp(2.0 * M_PI * i.x / float(C));
	W.y= exp(2.0 * M_PI * i.y / float(C));
	float2 Wm;
	Wm.x= L;
	Wm.y= L;
	float2 Wn;
	Wn.x= L;
	Wn.y= L;
	for (int m = 0; m < C; m++)
	{
		for (int n = 0; n < C; n++)
		{
			float2 denom;
			denom.x= 4.0;
			denom.y= 4.0;
			denom.x -= Wm.x + L / Wm.x + Wn.x + L / Wn.x;
			denom.y -= Wm.y + L / Wm.y + Wn.y + L / Wn.y;//se calcula el denominador para cada celda, segun el equema de discretizacion
			if (denom.x!= 0.0 && denom.y!= 0.0){
				calculoPoisson_h[m*C+n].x *= dx *dx / denom.x;
				calculoPoisson_h[m*C+n].y *= dx *dx / denom.y;
			}
			Wn.x *= W.x;//se multiplica por la constante W
			Wn.y *= W.y;
		}
		Wm.x *= W.x;
		Wm.y *= W.y;
	}
}


//////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void float2ToComplex(float2 *calculoPoisson_d, hipfftComplex *Phi_Poisson, int C){

  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if(i<C*C){
	  Phi_Poisson[i].x = calculoPoisson_d[i].x;
	  Phi_Poisson[i].y = calculoPoisson_d[i].y;
  }
}
//////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void ComplexToReal(hipfftComplex *T_I, float *poissonFinal_d, int C){

	int i = blockIdx.x*blockDim.x + threadIdx.x;
	  if(i<C*C){
		  poissonFinal_d[i] =T_I[i].x/float(1.e-6);

	  }
}
//////////////////////////////////////////////////////////////////////////////////////////////////////////
// Calculo campo electrico.

__global__ void ElectricBordes(float *poissonFinal_d, float *Ex, float *Ey, float L, int C) // recibe el potencial electroestatico calculado por la funcion poisson  y se calcula el campo electrico, tanto para X como para Y
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  float dx = L / float (C); // el delta de x representa el tamano de la malla

  if(i<C){
	  Ex[i*C]=(poissonFinal_d[((i+1)*C)-1] - poissonFinal_d[(i*C)+1])/(2. * dx);// hallando el campo en x, en la primera columna
	  Ex[((i+1)*C)-1] = (poissonFinal_d[((i+1)*C)-2] - poissonFinal_d[(i*C)]) / (2. * dx);// hallando el campo en x, en la ultima columna
	  Ey[((C-1)*C)+i] = (poissonFinal_d[((C-2)*C)+i] - poissonFinal_d[i]) / (2. * dx); //hallando el campo en "y" para la ultima fila
	  Ey[i] = (poissonFinal_d[((C-1)*C)+i] - poissonFinal_d[i+C]) / (2. * dx);//hallando el campo para la primera fila y la ultima
  }

}

__global__ void calculoCampoElectricoX(float *poissonFinal_d, float *Ex, float L, int C){
	 int i = blockIdx.x*blockDim.x + threadIdx.x;
	 int j = blockIdx.y*blockDim.y + threadIdx.y;
	 float dx = L / float (C); // el delta de x representa el tamano de la malla
	 if(i<C && j<C-2){
		 Ex[j+(C*i)] = (poissonFinal_d[j-1] - poissonFinal_d[j+1]) / (2. * dx);
	 }

}

__global__ void calculoCampoElectricoY(float *poissonFinal_d, float *Ey, float L, int C){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	float dx = L / float (C); // el delta de x representa el tamano de la malla
	if(i<((C*C)-C)){
		 Ey[i] = (poissonFinal_d[i-C] - poissonFinal_d[i+C]) / (2. * dx);

	}

}


//////////////////////////////////////////////////////////////////////////////////////////////////////////
int main(){
	// Parametros
	L = 64.0;            // dominio de la solucion 0 <= x <= L (en longitudes de debye)
	//L=LL*LL;
	N = 10000;            // Numero de particulas
	C = 64;            // Número de celdas.
	float vb = 3.0;    // velocidad promedio de los electrones
	//double kappa = 2. * M_PI / (L);
	//float dt=0.1;    // delta tiempo (en frecuencias inversas del plasma)
	//float tmax=10000;  // cantidad de iteraciones. deben ser 100 mil segun el material
	//int skip = int (tmax / dt) / 10; //saltos del algoritmo para reportar datos
	//int itera=0;
	 float salida=0.0;
	 float dx = L / float (C);

/////////////////////////////////////////////////////////////////////////////////////////////////////
//Inicializacion de la posición de las particulas en x, y y velocidad en vx,vy del host y dispositivo.
	float *rx_h,*ry_h,*vx_h,*vy_h;
	float *rx_d,*ry_d, *vx_d,*vy_d;
	int *jx_d, *jy_d;
	float *yx_d;
////////////////////////////////////////////////////////////////////////////////////////////////////
	// inicialización de las variables de densidad del host y dispositivo.
	float *ne_h;
	float *ne_d;
	float *n_h; // densidad normalizada.
	float *n_d; // densidad normalizada del dispositivo.
	float2 *calculoPoisson_h;
	float2 *calculoPoisson_d;
	float  * poissonFinal_h;
	float  * poissonFinal_d;
	float *Ex_h;
	float *Ey_h; //campoElectrico
	float *Ex_d;
	float *Ey_d; // Campo Electrico en el dispositivo.



////////////////////////////////////////////////////////////////////////////////////////////////////
	/*Crear la variable tipo cufftComplex*/
	hipfftComplex *n_d_C; // covertir la densidad en una variable compleja.
	hipfftComplex *T_F;	 // primer paso de la transformada hacia adelante
	hipfftComplex *T_F_N; //Trasformada hacia adelante normalizada.
	hipfftComplex *Phi_Poisson; // esta variable muestra la solucion de poisson.
	hipfftComplex *T_I;   // Transformada Inversa.

////////////////////////////////////////////////////////////////////////////////////////////////////
	int size = N*sizeof(float);
	int size_ne = C*C*sizeof(float);
	int size_ne2 = C*C*sizeof(float2);

//////////////////////////////////////////////////////////////////////////////////////////////////////
	//reserva en memoria al host
	rx_h = (float *)malloc(size);
	ry_h = (float *)malloc(size);
	vx_h = (float *)malloc(size);
	vy_h = (float *)malloc(size);
	ne_h = (float *)malloc(size_ne);
	n_h = (float *)malloc(size_ne);
	calculoPoisson_h=(float2 *)malloc(size_ne2);
	poissonFinal_h=(float *)malloc(size_ne);
	Ex_h = (float *)malloc(size_ne);
	Ey_h = (float *)malloc(size_ne);



//////////////////////////////////////////////////////////////////////////////////////////////////////
	//reserva de memoria del dispositivo.
	hipMalloc((void **)&rx_d,size);
	hipMalloc((void **)&ry_d,size);
	hipMalloc((void **)&vx_d,size);
	hipMalloc((void **)&vy_d,size);
	hipMalloc((void **)&ne_d,size_ne);
	hipMalloc((void **)&n_d,size_ne);
	hipMalloc((void **)&jx_d,size);
	hipMalloc((void **)&jy_d,size);
	hipMalloc((void **)&yx_d,size);
	hipMalloc((void **)&calculoPoisson_d,size_ne2);
	hipMalloc((void **)&poissonFinal_d,size_ne);
	hipMalloc((void **)&Ex_d,size_ne);
	hipMalloc((void **)&Ey_d,size_ne);


/////////////////////////////////////////////////////////////////////////////////////////////////////////
	/*Asignación de memoria a la variable tipo cufftComplex */
	hipMalloc((void **)&n_d_C,sizeof(hipfftComplex)*C*C);
	hipMalloc((void **)&T_F,sizeof(hipfftComplex)*C*C);
	hipMalloc((void **)&T_F_N,sizeof(hipfftComplex)*C*C);
	hipMalloc((void **)&Phi_Poisson,sizeof(hipfftComplex)*C*C);
	hipMalloc((void **)&T_I,sizeof(hipfftComplex)*C*C);
//////////////////////////////////////////////////////////////////////////////////////////////////////////

	//valores aleatorios y tamaños de los vectores.
	hiprandState *devStates;
	hipMalloc((void **) &devStates, N * sizeof(hiprandState));


	float blockSize = 1024;
	dim3 dimBlock (ceil(N/blockSize), 1, 1);
	dim3 dimBlock2 (ceil(C*C/blockSize), 1, 1);
	dim3 dimBlock3 (ceil(C*C/blockSize), ceil(C*C/blockSize), 1);
	dim3 dimGrid (blockSize, 1, 1);
	dim3 dimGrid3 (blockSize, blockSize, 1);
	int seed = time(NULL);


	distribucionParticulas<<<blockSize,dimBlock>>>(rx_d,ry_d,vx_d,vy_d,N,devStates,vb,L, seed);
	hipDeviceSynchronize();

	inicializacionDensidad<<<blockSize,dimBlock2>>>(ne_d,C);
	hipDeviceSynchronize();

	calculoDensidadInicializacionCeldas<<<blockSize,dimBlock>>>(rx_d,ry_d,jx_d,jy_d,yx_d,N,C,L);
	hipDeviceSynchronize();

	//funcion Calculo densidad.
	Output(ne_d,n_d,jx_d,jy_d,yx_d, C,L,N);     // Calculo de la densidad y normalización de la densidad.
	/////////////////////////////////////////////////////////////////////////////////////////////////////
	realTocomplex<<<blockSize,dimBlock2>>>(n_d, n_d_C,C);
	hipDeviceSynchronize();

	hipfftHandle plan;
	hipfftPlan2d(&plan,C,C,HIPFFT_C2C);
	hipfftExecC2C(plan,n_d_C,T_F,HIPFFT_FORWARD); // transformada hacia adelante en x and y.

	/*Valor de la transformada hacia adelante de latransformada rápida normalizada*/

	normalizacionSalidaTranfForward<<<blockSize,dimBlock2>>>(T_F,T_F_N, C);
	hipDeviceSynchronize();

	/*Calculo Poisson*/

	 ComplexToFloat2<<<blockSize,dimBlock2>>>(T_F_N,calculoPoisson_d,C);
	 hipDeviceSynchronize();

	 //Comprobacion del resultado de la transformada hacia adelante.
	 hipMemcpy(calculoPoisson_h, calculoPoisson_d, size_ne2, hipMemcpyDeviceToHost);

	 //Calculo Poisson antes de la transformada inversa
	 Poisson(calculoPoisson_h,L,C);

	 // Pasar el calculo de Poisson al dispositivo

	 hipMemcpy(calculoPoisson_d, calculoPoisson_h, size_ne2, hipMemcpyHostToDevice);

	 //Hacer la transformada Inversa
	 float2ToComplex<<<blockSize,dimBlock2>>>(calculoPoisson_d,Phi_Poisson,C);
	 hipDeviceSynchronize();

	 //Aplicar la transformada inversa de la matriz

	 hipfftExecC2C(plan,Phi_Poisson,T_I,HIPFFT_BACKWARD);

	 //tomando la transformada final.

	 ComplexToReal<<<blockSize,dimBlock2>>>(T_I,poissonFinal_d,C);
	 hipDeviceSynchronize();
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	 ElectricBordes<<<blockSize,dimBlock2>>>(poissonFinal_d,Ex_d,Ey_d, L,C); // Campo Electrico en los bordes.
	 hipDeviceSynchronize();

	 /*Calculo del campo electrico para x*/
	 calculoCampoElectricoX<<<dimGrid3,dimBlock3>>>(poissonFinal_d,Ex_d,L,C); // se utilizan dos hilos de debe organizar la manera como se envian.

	 /*Calculo del campo electrico para y*/
	 calculoCampoElectricoY<<<blockSize,dimBlock2>>>(poissonFinal_d, Ey_d,L,C);
	 hipDeviceSynchronize();




/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	//posicion en x.
	hipMemcpy(rx_h, rx_d, size, hipMemcpyDeviceToHost);

	// posicion en y.
	hipMemcpy(ry_h, ry_d, size, hipMemcpyDeviceToHost);

	// velocidad en x.
	hipMemcpy(vx_h, vx_d, size, hipMemcpyDeviceToHost);

	//velocidad en y.
	hipMemcpy(vy_h, vy_d, size, hipMemcpyDeviceToHost);
	//inicializacion densidades
	hipMemcpy(ne_h, ne_d, size_ne, hipMemcpyDeviceToHost);
	//normalización de la densidad.
	hipMemcpy(n_h, n_d, size_ne, hipMemcpyDeviceToHost);
	//calculo de la transformada rapida de fourier despues de la inversa.
	hipMemcpy(poissonFinal_h, poissonFinal_d, size_ne, hipMemcpyDeviceToHost);
	//Calculo de Campo Electrico Ex, Ey.
	hipMemcpy(Ex_h, Ex_d, size_ne, hipMemcpyDeviceToHost);
	hipMemcpy(Ey_h, Ey_d, size_ne, hipMemcpyDeviceToHost);

	/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

	ofstream init;
		init.open("distribucionInicial.txt");
		  		    for (int i = 0; i < N; i++){
		  		    	init<<rx_h[i]<<" "<<ry_h[i]<<" "<<vx_h[i]<<" "<<vy_h[i]<<endl;

		  		    }

		  		    init.close();


		init.open("salida_densidad3.txt");
					for (int i = 0; i < C*C; i++){
						init<<ne_h[i]<<" "<<endl;
						salida+=ne_h[i];
					}

					init.close();
					cout<<salida<<" "<<dx<<endl;

		init.open("densidadNormalizada.txt");
					for (int i = 0; i < C; i++){
						for (int j = 0; j < C; j++){
						init<<n_h[(i*C)+j]<<" ";
						}
					init<<endl;
					}

					init.close();
					cout<<salida<<" "<<dx<<endl;

	init.open("CalculoPoissonAntesdeLaTranformadaInversa.txt");
				for (int i = 0; i < C; i++){
					for (int j = 0; j < C; j++){
					init<<calculoPoisson_h[(i*C)+j].x<<" ";
					}
				init<<endl;
				}

				init.close();

	init.open("DespuesTransformadaInversaPoissonFinal");
					for (int i = 0; i < C; i++){
						for (int j = 0; j < C; j++){
						init<<poissonFinal_h[(i*C)+j]<<" ";
						}
					init<<endl;
					}

					init.close();


	init.open("CamposElectricos");

					for (int i = 0; i < C*C; i++){
						init<<Ex_h[i]<<" "<<Ey_h[i]<<endl;
					}

					init.close();

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
				/*Liberar memoria*/
	free(rx_h);
	free(ry_h);
	free(vx_h);
	free(vy_h);
	free(ne_h);
	free(n_h);
	free(calculoPoisson_h);
	free(poissonFinal_h);
	hipfftDestroy(plan);
	free(Ex_h);
	free(Ey_h);
	hipFree(rx_d);
	hipFree(ry_d);
	hipFree(vx_d);
	hipFree(vy_d);
	hipFree(ne_d);
	hipFree(n_d);
	hipFree(n_d_C);
	hipFree(T_F);
	hipFree(T_F_N);
	hipFree(Phi_Poisson);
	hipFree(T_I);
	hipFree(calculoPoisson_d);
	hipFree(poissonFinal_d);
	hipFree(Ex_d);
	hipFree(Ey_d);
	return (0);

}
