#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <math.h>
#include <time.h>
#include <fstream>
#include <vector>
#include <fstream>
#include <hiprand/hiprand_kernel.h>
#include <hipfft/hipfft.h>

float L,LL;

int N, C,itera;

using namespace std;

// función Maxwelliana de la distribución de las partículas.
__device__ float distribution (float vb, float aleatorio, hiprandState *states)     //generador de distribución maxwelliana para la velocidad
{

  // Genera un valor random v
   float fmax = 0.5 * (1.0 + exp (-2.0 * vb * vb));
   float vmin = - 5.0 * vb;
   float vmax = + 5.0 * vb;
   float v;
   float f;
   float x;
   int Idx = blockIdx.x*blockDim.x + threadIdx.x;

   while(true){
	   v = vmin + ((vmax - vmin) * aleatorio);
	   f = 0.5 * (exp (-(v - vb) * (v - vb) / 2.0) +
			    exp (-(v + vb) * (v + vb) / 2.0));
	   x = fmax * aleatorio;
	   if(x > f) aleatorio = hiprand_uniform(states + Idx);
	   else return v;
   }

 return 0;
}
//Distribución aleatoria de las partículas.
__global__ void distribucionParticulas(float *rx,float *ry,float *vx,float *vy,int N,hiprandState *states,float vb,float L, int seed){
	int Idx = blockIdx.x*blockDim.x + threadIdx.x;

	seed = (unsigned int) (clock() * Idx);
	hiprand_init(seed, 0, 0, states + Idx);

	if(Idx < N){
		rx[Idx] = L*hiprand_uniform(states + Idx);    //inicializando la posicion aleatoria en x
		ry[Idx] = L*hiprand_uniform(states + Idx);
		vx[Idx] = distribution(vb,hiprand_uniform(states + Idx),states);//;L*curand_uniform_float(states + Idx);//distribution(vb,states);                          //inicializa la velocidad con una distribucion maxwelliana
		vy[Idx] = distribution(vb,hiprand_uniform(states + Idx),states);//L*curand_uniform_float(states + Idx);//distribution(vb,states);                          //inicializa la velocidad con una distribucion maxwelliana

	}


}
// inicialización de la densidad.
__global__ void inicializacionDensidad(float *ne,int C){
	int Id=blockIdx.x*blockDim.x + threadIdx.x;
	if(Id<(C*C)){
		ne[Id] = 0.0;
	}
 }

__global__ void inicializacionValoresReales(float *vr,int C){
	int Id=blockIdx.x*blockDim.x + threadIdx.x;
	if(Id<(C*C)){
		vr[Id] = 0.0;
	}
 }

//Calculo de la densidad en cada celda.

__global__ void calculoDensidadInicializacionCeldas(float *rx, float *ry, int *jx,int *jy,float *yx, int N, int C,float L){
	int Id = blockIdx.x*blockDim.x + threadIdx.x;
	 float dx = L / float (C);
	 //float dxx = L /float(C*C);
	if(Id < N){
		 jx[Id] = int(rx[Id]/dx); //posicion en x de la particula
		 jy[Id] = int(ry[Id]/dx); //posicion en y de la particula
		 yx[Id] = (rx[Id]/dx) - (float)jx[Id]; //posicion exacta de la particula en x de la celda "j"
    }

}
__global__ void calculoDensidad(float *ne, int *jx, int *jy,float *yx, int C, float L, int N){
	 float dxx = L /float(C*C);
	 //int Id = blockIdx.x*blockDim.x + threadIdx.x;
	 for(int i=0; i<N; i++){
		ne[(jy[i]*C)+jx[i]] += (1. - yx[i])/dxx;
		if(jx[i]+1 == C) ne[(jy[i]*C)] += yx[i]/dxx;
		else ne[(jy[i]*C)+jx[i]+1] += yx[i]/dxx;
	 }

}

__global__ void normalizacionDensidad(float *ne,float *n, int N, int C, float L){
	 int Id = blockIdx.x*blockDim.x + threadIdx.x;
	 if(Id<C*C){
		 n[Id]=float(C*C)*ne[Id]/float(N)-1;
	 }

}


void Output(float *ne_d, float *n_d, int *jx_d,int *jy_d,float *yx_d,int C,float L,int N){

	float blockSize = 1024;
	dim3 dimBlock (ceil(N/blockSize), 1, 1);
	dim3 dimBlock2 (ceil(C*C/blockSize), 1, 1);
	dim3 dimGrid (blockSize, 1, 1);

	calculoDensidad<<<1,1>>>(ne_d,jx_d,jy_d,yx_d,C,L,N);//proceso de mejora.
	hipDeviceSynchronize();
	normalizacionDensidad<<<blockSize,dimBlock2>>>(ne_d,n_d,N,C,L);
	hipDeviceSynchronize();


}

//Asiganción inicial para el calculo de la transformada.

__global__ void realToComplex(float *n, hipfftComplex *n_C, int C){

  int Idx = blockIdx.x*blockDim.x + threadIdx.x;
  int Idy = blockIdx.y*blockDim.y + threadIdx.y;
  int index= Idx*C+Idy;
  if(Idx<C && Idy< C){
	  n_C[index].x=n[index];
	  n_C[index].y=0.0f;

  }
}
//Normalizacion de la salida de la transformada hacia adelante
__global__ void normalizaciónSalidaUt_F(hipfftComplex *n_C, hipfftComplex *uT_F, int C ){
	int Idx = blockIdx.x*blockDim.x + threadIdx.x;
	int Idy = blockIdx.y*blockDim.y + threadIdx.y;
	int index= Idx*C+Idy;
	  if(Idx<C && Idy< C){
		  uT_F[index].x=n_C[index].x/float(C*C*C*C);
		  uT_F[index].y=n_C[index].y/float(C*C*C*C);

	  }

}
// calculo de Poisson.
__global__ void calculoPoisson(hipfftComplex *uT_F, hipfftComplex *uC_P, float L, int C){
	uT_F[0.0].x =0.;

	float i(0.0, L); //creamos una variable compleja para poder aplicar la discretizacion.
	float W = exp(2.0 * M_PI * i / float (C));
	float Wm = L, Wn = L;
	float denom = 4.0;
	float dx=L/float(C-1); //delta x, el tamaño de la malla
	for (int i= 0; i< C; i+)
	{
		for (int j= 0; j< C; j++)
		{
			denom -= Wm + L / Wm + Wn + L / Wn; //se calcula el denominador para cada celda, segun el equema de discretizacion
			if (denom != 0.0){
				uT_F[i*C+j].x *= dx *dx / denom;
				uT_F[i*C+j].y *= dx *dx / denom;
			}
			Wn *= W;//se multiplica por la constante W
		}
		Wm *= W;
	}

	uT_F[0.0].x=0.;
	uT_F[0.0].y=0.;

	// asiganr a uC_P
}
__global__ void realToComplex( hipfftComplex *uT_I,float *poisson, int C){

  int Idx = blockIdx.x*blockDim.x + threadIdx.x;
  int Idy = blockIdx.y*blockDim.y + threadIdx.y;
  int index= Idx*C+Idy;
  if(Idx<C && Idy< C){
	  poisson_d[index]=uT_I[index].x/float(1.e+7);

  }
}





////////////////////////////////////////////////////////////////////////////////////////////////////
int main(){
	// Parametros
	L = 64.0;            // dominio de la solucion 0 <= x <= L (en longitudes de debye)
	//L=LL*LL;
	N = 10000;            // Numero de particulas
	C = 64;            // Número de celdas.
	float vb = 3.0;    // velocidad promedio de los electrones
	//double kappa = 2. * M_PI / (L);
	//float dt=0.1;    // delta tiempo (en frecuencias inversas del plasma)
	//float tmax=10000;  // cantidad de iteraciones. deben ser 100 mil segun el material
	//int skip = int (tmax / dt) / 10; //saltos del algoritmo para reportar datos
	//int itera=0;
	 float salida=0.0;
	 float dx = L / float (C);

/////////////////////////////////////////////////////////////////////////////////////////////////////
//Declaración de la posición de las particulas en x, y y velocidad en vx,vy del host y dispositivo.
	float *rx_h,*ry_h,*vx_h,*vy_h;
	float *rx_d,*ry_d, *vx_d,*vy_d;
	int *jx_d, *jy_d;
	float *yx_d;
////////////////////////////////////////////////////////////////////////////////////////////////////
	// Declaración de las variables de densidad del host y dispositivo.
	float *ne_h;
	float *ne_d;
	float *n_h;
	float *n_d;
	float *poisson_h;
	float *poisson_d;
///////////////////////////////////////////////////////////////////////////////////////////////////
	// Declaración de las variables tipo complejas con hipfftComplex.
	hipfftComplex *n_C, *uT_F,*uC_P,*uT_I; // n_C= Densidad Compleja. uT_F=Salida transformada rapida
									// uC_P= Salida cälculo Poisson.

////////////////////////////////////////////////////////////////////////////////////////////////////
	int size = N*sizeof(float);
	int size_ne = C*C*sizeof(float);

//////////////////////////////////////////////////////////////////////////////////////////////////////
	//reserva en memoria al host
	rx_h = (float *)malloc(size);
	ry_h = (float *)malloc(size);
	vx_h = (float *)malloc(size);
	vy_h = (float *)malloc(size);
	ne_h = (float *)malloc(size_ne);
    n_h  = (float *)malloc(size_ne);
    poisson_h  = (float *)malloc(size_ne);
//////////////////////////////////////////////////////////////////////////////////////////////////////
	//reserva de memoria del dispositivo.
	hipMalloc((void **)&rx_d,size);
	hipMalloc((void **)&ry_d,size);
	hipMalloc((void **)&vx_d,size);
	hipMalloc((void **)&vy_d,size);
	hipMalloc((void **)&ne_d,size_ne);
	hipMalloc((void **)&n_d,size_ne);
	hipMalloc((void **)&jx_d,size);
	hipMalloc((void **)&jy_d,size);
	hipMalloc((void **)&yx_d,size);
	hipMalloc((void **)&poisson_d,size_ne);
///////////////////////////////////////////////////////////////////////////////////////////////////////
	//reserva memoria sobre variables complejas.
	hipMalloc((void **)&n_C,size_ne);
	hipMalloc((void **)&uT_F,size_ne);
	hipMalloc((void **)&uC_P,size_ne);
	hipMalloc((void **)&uT_I,size_ne);

///////////////////////////////////////////////////////////////////////////////////////////////////////

	//valores aleatorios y tamaños de los vectores.
	hiprandState *devStates;
	hipMalloc((void **) &devStates, N * sizeof(hiprandState));


	float blockSize = 1024;
	dim3 dimBlock (ceil(N/blockSize), 1, 1);
	dim3 dimBlock2 (ceil(C*C/blockSize), 1, 1);
	dim3 dimGrid (blockSize, 1, 1);
	int seed = time(NULL);


	distribucionParticulas<<<blockSize,dimBlock>>>(rx_d,ry_d,vx_d,vy_d,N,devStates,vb,L, seed);
	hipDeviceSynchronize();

	inicializacionDensidad<<<blockSize,dimBlock2>>>(ne_d,C);
	hipDeviceSynchronize();

	calculoDensidadInicializacionCeldas<<<blockSize,dimBlock>>>(rx_d,ry_d,jx_d,jy_d,yx_d,N,C,L);
	hipDeviceSynchronize();
//
//	calculoDensidad<<<1,1>>>(ne_d,jx_d,jy_d,yx_d,C,L,N);//proceso de mejora.
//	hipDeviceSynchronize();
//////////////////////////////////////////////////////////////////////////////////////////////////////////////

	//funcion Calculo densidad.


	Output(ne_d,n_d,jx_d,jy_d,yx_d, C,L,N);// Calculo de la densidad y normalización de la densidad.



/////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	//calculo de Poisson.
	realToComplex<<<blockSize,dimBlock2>>>(n_d,n_C,C);
	hipfftHandle plan;
	hipfftPlan2d(&plan,C,C,HIPFFT_C2C);
	hipfftExecC2C(plan,n_C,uT_F,HIPFFT_FORWARD);
	normalizaciónSalidaUt_F<<<blockSize,dimBlock2>>>(n_C,uT_F,C);
	//calculoPoisson<<<blockSize,dimBlock2>>>(uT_F,uC_P, L, C);
	hipfftExecC2C(plan,uC_P,uT_I,HIPFFT_BACKWARD);
	realToComplex<<<blockSize,dimBlock2>>>( uT_I,poisson_d,C);

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	//Asignar los datos del dispositivo al host.
	//posición en x.
	hipMemcpy(rx_h, rx_d, size, hipMemcpyDeviceToHost);

	// posición en y.
	hipMemcpy(ry_h, ry_d, size, hipMemcpyDeviceToHost);

	// velocidad en x.
	hipMemcpy(vx_h, vx_d, size, hipMemcpyDeviceToHost);

	//velocidad en y.
	hipMemcpy(vy_h, vy_d, size, hipMemcpyDeviceToHost);
	//inicializacion densidades
	hipMemcpy(ne_h, ne_d, size_ne, hipMemcpyDeviceToHost);
	//NormalizacionDensidades
	hipMemcpy(n_h, n_d, size_ne, hipMemcpyDeviceToHost);
	// Calculo de Poisson
	hipMemcpy(poisson_h, poisson_d, size_ne, hipMemcpyDeviceToHost);


/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

	//Imprimir resultados.

	ofstream init;
		init.open("distribucionInicial.txt");
		  		    for (int i = 0; i < N; i++){
		  		    	init<<rx_h[i]<<" "<<ry_h[i]<<" "<<vx_h[i]<<" "<<vy_h[i]<<endl;

		  		    }

		  		    init.close();


		init.open("salida_densidad3.txt");
					for (int i = 0; i < C*C; i++){
						init<<ne_h[i]<<" "<<endl;
						salida+=ne_h[i];
					}

					init.close();
					cout<<salida<<" "<<dx<<endl;

		init.open("densidadNormalizada.txt");
					for (int i = 0; i < C; i++){
						for (int j = 0; j < C; j++){
						init<<n_h[(i*C)+j]<<" ";
						}
					init<<endl;
					}

					init.close();
					cout<<salida<<" "<<dx<<endl;



///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	//Liberar memoria.
	free(rx_h);
	free(ry_h);
	free(vx_h);
	free(vy_h);
	free(ne_h);
	free(n_h);
	free(poisson_h);
	hipfftDestroy(plan);
	hipFree(rx_d);
	hipFree(ry_d);
	hipFree(vx_d);
	hipFree(vy_d);
	hipFree(ne_d);
	hipFree(n_d);
	hipFree(n_C);
	hipFree(uT_F);
	hipFree(uC_P);
	hipFree(uT_I);
	hipFree(poisson_d);

	return (0);

}
