/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <math.h>
#include <time.h>
#include <fstream>
#include <vector>
#include <fstream>
#include <hiprand/hiprand_kernel.h>

float L,LL; int N, C,itera;

using namespace std;


__device__ float distribution (float vb, float aleatorio, hiprandState *states)     //generador de distribución maxwelliana para la velocidad
{

  // Genera un valor random v
   float fmax = 0.5 * (1.0 + exp (-2.0 * vb * vb));
   float vmin = - 5.0 * vb;
   float vmax = + 5.0 * vb;
   float v;
   float f;
   float x;
   int Idx = blockIdx.x*blockDim.x + threadIdx.x;

   while(true){
	   v = vmin + ((vmax - vmin) * aleatorio);
	   f = 0.5 * (exp (-(v - vb) * (v - vb) / 2.0) +
			    exp (-(v + vb) * (v + vb) / 2.0));
	   x = fmax * aleatorio;
	   if(x > f) aleatorio = hiprand_uniform(states + Idx);
	   else return v;
   }

}

__global__ void distribucionParticulas(float *rx,float *ry,float *vx,float *vy,int N,hiprandState *states,float vb,float L){
	int Idx = blockIdx.x*blockDim.x + threadIdx.x;

	unsigned int seed = (unsigned int) (clock() * Idx);
	hiprand_init(seed, 0, 0, states + Idx);

	if(Idx < N){
		 rx[Idx] = L*hiprand_uniform(states + Idx);    //inicializando la posicion aleatoria en x
		 ry[Idx] = L*hiprand_uniform(states + Idx);
		 vx[Idx] = distribution(vb,hiprand_uniform(states + Idx),states);//;L*curand_uniform_float(states + Idx);//distribution(vb,states);                          //inicializa la velocidad con una distribucion maxwelliana
		 vy[Idx] = distribution(vb,hiprand_uniform(states + Idx),states);//L*curand_uniform_float(states + Idx);//distribution(vb,states);                          //inicializa la velocidad con una distribucion maxwelliana

	}

}


int main(){
	// Parametros
	L = 25.0;            // dominio de la solucion 0 <= x <= L (en longitudes de debye)
	//L=LL*LL;
	N = 10000;            // Numero de particulas
	C = 50;          // Numero de celdas EN UNA DIMENSION, EL TOTAL DE CELDAS ES C*C
	float vb = 3.0;    // velocidad rayo promedio
	//float dt=0.1;    // delta tiempo (en frecuencias inversas del plasma)
	//float tmax=10000;  // cantidad de iteraciones. deben ser 100 mil segun el material
	//int skip = int (tmax / dt) / 10; //saltos del algoritmo para reportar datos
	//int itera=0;
	float *rx_h,*ry_h,*vx_h,*vy_h;
	float *rx_d,*ry_d,*vx_d,*vy_d;

	int size = N*sizeof(float);
	//reserva en memoria al host
	rx_h = (float *)malloc(size);
	ry_h = (float *)malloc(size);
	vx_h = (float *)malloc(size);
	vy_h = (float *)malloc(size);
	//reserva de memoria del dispositivo.
	hipMalloc((void **)&rx_d,size);
	hipMalloc((void **)&ry_d,size);
	hipMalloc((void **)&vx_d,size);
	hipMalloc((void **)&vy_d,size);
	//valores aleatorios.
	hiprandState *devStates;
	hipMalloc((void **) &devStates, N * sizeof(hiprandState));

	//lanzar el kernel. El primer parámetro que va al llamarse un kernel es la cantidad de hilos que queremos que haya en cada bloque.
	//despues va la cantidad de bloques

	float blockSize = 1024;
	dim3 dimBlock (ceil(N/blockSize), 1, 1);
	dim3 dimGrid (blockSize, 1, 1);


	distribucionParticulas<<<blockSize,dimBlock>>>(rx_d,ry_d,vx_d,vy_d,N,devStates,vb,L);
	// ontener los resultados.
	//posición en x.
	hipMemcpy(rx_h, rx_d, size, hipMemcpyDeviceToHost);

	// posición en y.
	hipMemcpy(ry_h, ry_d, size, hipMemcpyDeviceToHost);

	// velocidad en x.
	hipMemcpy(vx_h, vx_d, size, hipMemcpyDeviceToHost);

	//velocidad en y.
	hipMemcpy(vy_h, vy_d, size, hipMemcpyDeviceToHost);

	//Imprimir el resultado
	for(int i = 0; i < N ;i++){
		printf("%f %f %f %f\n",rx_h[i],ry_h[i],vx_h[i],vy_h[i]);
	}

	free(rx_h);
	free(ry_h);
	free(vx_h);
	free(vy_h);
	hipFree(rx_d);
	hipFree(ry_d);
	hipFree(vx_d);
	hipFree(vy_d);

	return (0);

}
